#include "hip/hip_runtime.h"
#include "tsp.h"

#include <stdio.h>
#include <limits.h>






/*
 *   Function reduce_matrix_row : 
 *
 *   inputs : augmented adjacent matrix and size  
 *
 *   outputs: reduced matrix given , row and column , selection 
 *
 *
 */

__global__ void reduce_matrix_row( int *aug_adj_mat ,int offset ,  int row , int col,  int size , int dim)
{

  
     int count  = 0 ;    	
     int span_size = gridDim.x * blockDim.x; 
     int tid =   threadIdx.x; 
     int min_so_far;
     int iter = 0 ; 
     int val; 
     int total_count = size / span_size + 1; 
     int *mat = (aug_adj_mat + offset); 
     if ( (row >= 0) && (col >= 0 ) )
     {
              //*choice_val = get_i_j(mat,row,col,dim);
	      set_i_j(&mat,col,row,dim,2*INFINITE);

	    
	     for ( iter = 0 ; iter  < size ; iter++)
	     {

		    set_i_j(&mat,iter,col,dim,2*INFINITE);
		    set_i_j(&mat,row,iter,dim,2*INFINITE);
	            set_i_j(&mat,iter,iter,dim,2*INFINITE);
                    
 	      

	     }
     }
     for ( count = 0 ; count < total_count ; count++)
     {

         tid = threadIdx.x + count * span_size + blockIdx.x * blockDim.x; 

	 if ( tid < size ) 
	 {
                     min_so_far = get_i_j(mat, tid , 0 , dim ) ; 
		     for ( iter = 1 ; iter < size ; iter++)
		     {
		            min_so_far =   minimum(get_i_j(mat,tid, iter,dim) , min_so_far);
		     
		     }

		     // if infinity then ignore 
		     if ( min_so_far >= INFINITE )
		     {

			   min_so_far =  0;    
		     }
		     set_i_j(&mat,tid,size,dim,min_so_far); 
                     for ( iter  = 0 ; iter < size ; iter++)
		     {
			  val = get_i_j ( (mat), tid, iter , dim); 
			  
			  if ( val <=  INFINITE )
			     set_i_j(&mat, tid, iter, dim, val - min_so_far); 

		      
		     }
		     
                   
	 }

     
     }


}	
/*
 *   Function reduce_matrix_col : 
 *
 *   inputs : augmented adjacent matrix and size  
 *
 *   outputs: reduced matrix given , row and column , selection 
 *
 *
 */


__global__ void reduce_matrix_col( int *aug_adj_mat,int offset, int row , int col,  int size , int dim )
{

  
     int count  = 0 ;    	
     int span_size = gridDim.x * blockDim.x; 
     int tid =   threadIdx.x; 
     int min_so_far;
     int iter = 0 ; 
     int val; 
     int total_count = size / span_size + 1;
     
     int *mat = (aug_adj_mat + offset); 

     for ( count = 0 ; count < total_count ; count++)
     {

         tid = threadIdx.x + count * span_size + blockIdx.x * blockDim.x; 

	 if ( tid < size ) 
	 {
                     min_so_far = get_i_j(mat,0 , tid , dim ) ; 
		     for ( iter = 1 ; iter < size ; iter++)
		     {
		            min_so_far =   minimum(get_i_j(mat,iter, tid,dim) , min_so_far);
		     
		     }
		      // if infinity then ignore 
		     if ( min_so_far >= INFINITE )
		     {

			   min_so_far =  0;    
		     }
		     
		    
		     set_i_j(&mat,size,tid,dim,min_so_far); 
                     for ( iter  = 0 ; iter < size ; iter++)
		     {
			  val = get_i_j (  mat, iter, tid , dim); 
			  if ( val <= INFINITE ) 
			      set_i_j(&mat , iter, tid,  dim, val - min_so_far); 

		      
		     }
                   
	 }

     
     }




}	


/*
 *   Function calc_lower_bound: 
 *
 *   inputs : reduced matrix , size 
 *     
 *
 *
 */


__global__ void calc_lower_bound( int *aug_adj_matx , int offset, int parent_offset, int row , int col , int size,  int dim)
{

    __shared__ int  tmp_buf[SHARED_MEM_SIZE]; 
    
     int tid = threadIdx.x + blockIdx.x * gridDim.x ; 
     int span_size = gridDim.x * blockDim.x; 
     int total_count = ( (dim-1) / span_size ) + 1 ; 
     int iter = 0 ; 
     int *red_matrix = aug_adj_matx +   offset  * dim * dim ; 
     int *par_matrix = aug_adj_matx + parent_offset * dim * dim ; 
     int lower_bound = 0 ; 
     int lbc1 = 0 , lbc2 = 0 ; 
      // initialized the shared memory buffer to  0 

     for ( iter = threadIdx.x ; iter < SHARED_MEM_SIZE ; iter += blockDim.x )
     {
                
        tmp_buf[iter] = 0 ; 
	__syncthreads(); 
     }

     for ( iter = 0 ; iter < total_count ; iter++)
     {
          if ( tid  < (size) ) 
	  {
	                
		       tmp_buf[threadIdx.x ] += get_i_j(red_matrix , tid ,size ,dim)  + get_i_j(red_matrix,size ,tid , dim); 
		       tid += span_size;  
		       __syncthreads(); 	
	  }
        	
               
     }  


     for ( iter =  blockDim.x/2 ; iter > 0 ; iter >>=1)
     {

            if ( threadIdx.x < iter ) 
	    {

	            tmp_buf[threadIdx.x] +=  tmp_buf[threadIdx.x + iter]; 
	    }
	    __syncthreads(); 

     }

     if ( threadIdx.x  == 0 )
     {

	     
	   lbc1 =  get_i_j ( par_matrix , row , col , dim);
	   lbc2 =  get_i_j ( par_matrix , size , size ,dim) ; 
	   if ( lbc1 >= INFINITE  || lbc2 >= INFINITE  )
	   {
                   lower_bound = INFINITE; 
	   } 
	   else 
	   {
                   lower_bound = lbc1 + lbc2 + tmp_buf[0];  

	   }
			   
           set_i_j(&red_matrix , size,size , dim , lower_bound);   

     }


}

__global__ void copy_matrix ( int *dest , int*src ,  int dim )
{


     int count  = 0 ;    	
     int span_size = gridDim.x * blockDim.x; 
     int tid =   threadIdx.x; 
     int iter = 0 ; 
     int total_count = dim / span_size + 1;   
     for ( count = 0 ; count < total_count ; count++)
     {

         tid = threadIdx.x + count * span_size + blockIdx.x * blockDim.x; 

	 if ( tid < dim ) 
	 {
		     for ( iter = 0 ; iter < dim ; iter++)
		     {
		             set_i_j ( &dest , tid, iter,dim, get_i_j(src,iter, tid,dim) ) ;
		     
		     }
		     
                   
	 }

     
     }

}



/*
 *   Function: get_i_j  
 *     
 *   Description : Utility function for converting a 2d array into a id array , given the row and column , returns the value at row , col 
 *   
 *
 *
 */

__device__ inline  int get_i_j( int *matrix , int row , int col , int dim)
{ 


	return matrix[row*dim+col]; 
}	

/*   
 *  Function : set_i_j  int *
 *
 *  Description: Given a matrix , row and column, dimension and value , fixes a value at i , j  
 *  
 *  
 *
 */

__device__  inline void set_i_j ( int **matrix , int row , int col , int dim, int val)
{


	(*matrix)[row*dim + col] =val; 
}	

/*__device__ int choice_val; 

 *
 *   Function: minimum 
 *
 *   Description : Given 2 number a , b returns the minuimum of the 2 
 *
 *
 *
 *
 */

__device__ inline int minimum( int val1 ,int  val2)
{ 


         if ( val1 < val2) 
		 return val1;
	 return val2; 



}	



