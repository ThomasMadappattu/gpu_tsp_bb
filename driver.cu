#include <stdio.h>
#include <stdlib.h> 

#include "tsp.h"
#include "queue.h" 


#define KILOB  (1024) 
#define MEGB  (KILOB * KILOB ) 
#define GIGA  ( KILOB * MEGB )  


enum
{
   FALSE,
   TRUE

};


/* Allocation megabytes on GPU */ 
#define  GPU_ALLOC  ( 512 * MEGB)


// remove this remove test code 
#define TEST 

void allocate_adj_matrix( int **mat , char *filename, int dim, int mat_size ); 
void copy_matrix_to_device ( int *matrix , int dim , int **cuda_mat); 
void copy_matrix_to_host( int **matrix , int dim ,int *cuda_mat); 
void dump_matrix ( int *mat , int size) ;

void copy_matrix_to_offset( int start_offset , int dest_offset , int dim ); 
int  extract_lower_bound ( int *mat ,int size ,  int dim ); 

void solve_tsp(PRIOR_QUEUE_T queue , int *adj_matrix , int dim);

inline void evaluate_path(int *parent,int *child  , PRIOR_QUEUE_T queue , int dim); 
void solve_tsp2(PRIOR_QUEUE_T queue , int *adj_matrix , int dim);

int is_visted( int *path, int len , int vertex);

/*
     globals 

*/

int __current_allocation = 0 ; 
int *__cudaMatrix; 


int main()
{


        int *matrix; 
        hipError_t err; 
        PRIOR_QUEUE_T queue;   
        int *dev_mat_ptr;
	int lb; 
	int *dev_ptr; 

        // Allocat a huge array  in GPU for our matrix 

        err = hipMalloc( (void**)&__cudaMatrix , GPU_ALLOC);


        if ( err != hipSuccess )
        {
            fprintf(stderr,"\n Unable to allocate cuda  ( GPU men for our matrix ) memory !");
	    exit(EXIT_FAILURE); 
    
    
        }
        init_prior_queue(&queue);    	
  



	// Define a test variable at the top , for testing .. 

        #ifdef TEST    	
	     init_prior_queue(&queue); 
	     allocate_adj_matrix(&matrix,(char *)"test.txt",5,4);
             dump_matrix(matrix,5); 
             copy_matrix_to_device(matrix , 5 , &__cudaMatrix ); 
             reduce_matrix_row<<<1, 1>>>(__cudaMatrix,0,1,2,4,5); 
             reduce_matrix_col<<<1, 1>>>(__cudaMatrix,0,1,2,4,5); 	 
             calc_lower_bound<<<1,1024>>>(__cudaMatrix, 0 ,__cudaMatrix, 0 , 1 , 2 , 4 ,  5 ) ; 
	     lb = extract_lower_bound(__cudaMatrix,4,5); 
             printf( " \n lower bound = %d" , lb); 
	     hipMalloc( (void **)&dev_ptr , 25* sizeof(int)); 
             copy_matrix<<<10,10>>>(dev_ptr,__cudaMatrix, 5);      
	     copy_matrix_to_host(&matrix , 5, dev_ptr);
             hipDeviceSynchronize();
	     printf("\n Reduced Matrix ... \n"); 
	     dump_matrix(matrix,5);
	     
	     allocate_adj_matrix(&matrix,(char *)"test.txt",5,4);
	     solve_tsp2(queue,matrix,5);
            #endif 	     
        
	     
         free(matrix); 
}



void allocate_adj_matrix ( int **mat , char *filename , int dim , int mat_size)
{

    int *adj_matrix = (int *) malloc(sizeof(int) * dim * dim ); 
    
    FILE *fp = fopen ( filename , "r");
    memset(adj_matrix , 0 , sizeof(int) * dim * dim ); 
    if ( fp == NULL  ) 
    {
          fprintf(stderr, " Unable to open the file !");
	  exit(EXIT_FAILURE);

    }
    int row , col ; 
    for ( row = 0 ; row < mat_size ; row++)
    {
	    for ( col = 0 ; col < mat_size ; col++)
	    {

	            fscanf(fp, "%d", &adj_matrix[row*dim + col]) ;  
	    
	    
	    }


    }
    *mat = adj_matrix; 
    fclose(fp); 
      

}

void dump_matrix ( int *mat , int dim)
{
    int row , col ; 	
   for ( row = 0 ; row < dim ; row++)
   {
       for ( col = 0 ; col < dim ; col++)
       {
 
            printf("%d\t\t\t" , mat[row*dim + col]);        
      
      
       }
       printf("\n"); 
  
  
   } 


}

void copy_matrix_to_device ( int *matrix , int dim , int **cuda_mat)
{
    hipError_t err = hipMalloc( (void**)cuda_mat , sizeof(int) * dim * dim);
    hipEvent_t sync;
    hipEventCreate(&sync); 
    if ( err != hipSuccess )
    {
         fprintf(stderr,"\n Unable to allocate cuda memory !");
	 exit(EXIT_FAILURE); 
    
    
    }
    hipMemset(*cuda_mat,0,dim*dim);
    hipMemcpy(*cuda_mat,matrix , dim * dim * sizeof(int) , hipMemcpyHostToDevice);  
    hipEventRecord(sync,0);

}

void copy_matrix_to_host( int **matrix , int dim , int *cuda_mat)
{
    
    hipEvent_t sync; 
    hipEventCreate(&sync);
    hipMemcpy(*matrix,cuda_mat , dim * dim * sizeof(int) , hipMemcpyDeviceToHost);  
    hipEventRecord(sync,0);
    hipEventSynchronize(sync);       


}


int  extract_lower_bound ( int *mat ,int size ,  int dim )
{
        
     	int lower_bound; 
           
       hipEvent_t sync; 
       hipEventCreate(&sync);

      hipMemcpy(&lower_bound ,   (mat   + size * dim + size) , sizeof(lower_bound) , hipMemcpyDeviceToHost ); 	      
      hipEventRecord(sync,0);
      hipEventSynchronize(sync);       
      return lower_bound ; 

}	






void evaluate_path(int *parent,int *child  , PRIOR_QUEUE_T queue , int dim)
{

       
      int level =  1; 
      int temp;          	
      int vertex  = 0 ; 
      PRIOR_QUEUE_T new_item,top=NULL;
      init_prior_queue(&new_item);  
      init_prior_queue(&top);  
      int parent_lb = 0;
      int *path_so_far;
      int level_so_far;
      int lower_bound_so_far;
      path_so_far= (int *)malloc(sizeof(int) * (dim + 1) ); 
      level_so_far = queue->level ;       
      lower_bound_so_far = queue->lower_bound;
      memcpy(path_so_far,queue->path , sizeof(int) *( dim + 1));
     
       
     for ( level = 1 ; level < queue->level ; level++)
     {
    
	     
    	   
	    reduce_matrix_row<<<10,1024>>>(child,0, queue->path[level-1] , queue->path[level] ,dim-1, dim);
            reduce_matrix_col<<<10,1024>>>(child,0, queue->path[level-1] , queue->path[level] ,dim-1, dim);


     }	    

     delete_prior_queue(&top,&queue);

    
     copy_matrix<<<10,1024>>>(parent, child, dim);    

       
     for ( vertex = 0 ; vertex < dim ; vertex++)
      {
         
	 if ( vertex != path_so_far[level_so_far - 1])
	 {	 
		 new_item = allocate_prior_queue();
		 new_item->path = (int *)malloc(sizeof(int) * (dim + 1) ) ; 
		 memcpy(new_item->path,path_so_far, sizeof(int) *( dim + 1));
		 new_item->level = level_so_far; 
		 add_path(vertex , &new_item, dim);
		 reduce_matrix_row<<<10,1024>>>(child,0, new_item->path[new_item->level-1] , new_item->path[new_item->level-1] ,dim-1, dim);
		 reduce_matrix_col<<<10,1024>>>(child,0, new_item->path[new_item->level-1] , new_item->path[new_item->level-1] ,dim-1, dim);
		 calc_lower_bound<<<1,1024>>>(child, 0 ,parent, 0 ,  new_item->path[new_item->level-1] , new_item->path[new_item->level], dim-1,dim); 
		 hipDeviceSynchronize(); 
		 new_item->lower_bound = extract_lower_bound(child,dim-1,dim) +  lower_bound_so_far; 	 
	         insert_prior_queue(new_item,&queue);


	}
      } 
   printf(" \n Evaluate --- "); 

   dump_queue(queue);
}

int is_visted( int *path, int len , int vertex)
{

    int count = 0 ;
    for ( count = 0 ; count < len; count++)
    {

        if ( path[count] == vertex )
		return TRUE;
    
    }

    return FALSE; 
}



void solve_tsp2(PRIOR_QUEUE_T queue , int *adj_matrix , int dim)
{


         int *cu_child , *cu_parent,  *cu_red; 
         PRIOR_QUEUE_T new_item,top; 
         int temp;
	 int done = 0;

         int level =  1; 
         int vertex  = 0 ; 
         init_prior_queue(&new_item);  
         init_prior_queue(&top);  
         int parent_lb = 0;
         int *path_so_far;
         int level_so_far;
         int lower_bound_so_far;
         int temp_lb;       

	 copy_matrix_to_device(adj_matrix , dim , &cu_parent); 
         copy_matrix_to_device(adj_matrix, dim, &cu_child);    	 
         copy_matrix_to_device(adj_matrix,dim, &cu_red);

	 init_prior_queue(&new_item); 
  
	 // Calculate reduced matrix  
	 reduce_matrix_row<<<10,1024>>>(cu_child,0, -1 , -1 ,dim-1, dim);
         reduce_matrix_col<<<10,1024>>>(cu_child,0, -1,  -1 ,dim-1, dim);
	 calc_lower_bound<<<1,1024>>>(cu_child, 0 ,cu_parent, 0 ,  dim-1 , dim-1 , dim-1,dim); 
         copy_matrix<<<10,1024>>>(cu_red,cu_child, dim); 
	 new_item = allocate_prior_queue(); 
         new_item->path = (int *)malloc(sizeof(int) * ( dim  + 1) ) ; 
	 new_item->level = 0;
	 new_item->lower_bound =  extract_lower_bound(cu_child,dim-1,dim);  
         add_path(0,&new_item,dim); 
         insert_prior_queue(new_item , &queue); 	 
             
	 while(!done )
	 {

                
	      path_so_far= (int *)malloc(sizeof(int) * (dim + 1) ); 
	      level_so_far = queue->level ;       
	      lower_bound_so_far = queue->lower_bound;
	      memcpy(path_so_far,queue->path , sizeof(int) *( dim + 1));
	     
	     copy_matrix<<<10,1024>>>(cu_child,cu_red,dim); 
	     for ( level = 1 ; level < queue->level ; level++)
	     {
	    
		     
		   
		    reduce_matrix_row<<<10,1024>>>(cu_child,0, queue->path[level-1] , queue->path[level] ,dim-1, dim);
		    reduce_matrix_col<<<10,1024>>>(cu_child,0, queue->path[level-1] , queue->path[level] ,dim-1, dim);


	     }	    

	     delete_prior_queue(&top,&queue);

	    
	     copy_matrix<<<10,1024>>>(cu_parent,cu_child, dim);    

	       
	     for ( vertex = 0 ; vertex < dim-1 ; vertex++)
	      {
		 
		 if (is_visted(path_so_far,level_so_far,vertex)==FALSE)
		 {	 
			 new_item = allocate_prior_queue();
			 new_item->path = (int *)malloc(sizeof(int) * (dim + 1) ) ; 
			 memcpy(new_item->path,path_so_far, sizeof(int) *( dim + 1));
			 new_item->level = level_so_far; 
			 add_path(vertex , &new_item, dim);
			 reduce_matrix_row<<<10,1024>>>(cu_child,0, new_item->path[new_item->level-1] , new_item->path[new_item->level-1] ,dim-1, dim);
			 reduce_matrix_col<<<10,1024>>>(cu_child,0, new_item->path[new_item->level-1] , new_item->path[new_item->level-1] ,dim-1, dim);
			 calc_lower_bound<<<1,1024>>>(cu_child, 0 ,cu_parent, 0 ,  new_item->path[new_item->level-1] , new_item->path[new_item->level], dim-1,dim); 
			 hipDeviceSynchronize(); 
		
		         temp_lb =  extract_lower_bound(cu_child,dim-1,dim) ;
			 if( temp_lb >= INFINITE || lower_bound_so_far >= INFINITE ) 
			      new_item->lower_bound = INFINITE; 
		         else

			 new_item->lower_bound = temp_lb  +  lower_bound_so_far; 	 
			
		         insert_prior_queue(new_item,&queue);


		}
	      }       
	     dump_queue(queue);
	     if ( queue == NULL ) 
			break;
	     if ( queue->level == dim-1)
		break;

	 }


	dump_queue(queue);
 	


}




void solve_tsp(PRIOR_QUEUE_T queue , int *adj_matrix , int dim)
{

         int *cu_child , *cu_parent; 
         PRIOR_QUEUE_T new_item,top; 
         int temp;
	 int done = 0;
	 copy_matrix_to_device(adj_matrix , dim , &cu_parent); 
         copy_matrix_to_device(adj_matrix, dim, &cu_child);    	 
         init_prior_queue(&new_item); 
  
	 // Calculate reduced matrix  
	 reduce_matrix_row<<<10,1024>>>(cu_child,0, -1 , -1 ,dim-1, dim);
         reduce_matrix_col<<<10,1024>>>(cu_child,0, -1,  -1 ,dim-1, dim);
	 calc_lower_bound<<<1,1024>>>(cu_child, 0 ,cu_parent, 0 ,  dim-1 , dim-1 , dim-1,dim); 

	 new_item = allocate_prior_queue(); 
         new_item->path = (int *)malloc(sizeof(int) * ( dim  + 1) ) ; 
	 new_item->level = 0;
	 new_item->lower_bound =  extract_lower_bound(cu_child,dim-1,dim);  
         add_path(0,&new_item,dim); 
         insert_prior_queue(new_item , &queue); 	 
         while(!done )
	 {

                evaluate_path(cu_parent , cu_child, queue , dim );
	       
               	dump_queue(queue);
		if ( queue == NULL ) 
			break;
		if ( queue->level == dim-1)
			break;

	 }


	dump_queue(queue);
 	 

}
